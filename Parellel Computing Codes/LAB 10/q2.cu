#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void rowPower(int *dArr, int m, int n) {
    int row = (blockIdx.x / n) + 1;
    int mul = 1;
    while(row > 0) {
        mul *= dArr[blockIdx.x];
        row--;
    }
    dArr[blockIdx.x] = mul;
}

int main() {
    int m, n;
    printf("Enter the dimensions of the matrix: ");
    scanf("%d %d", &m, &n);
    int arr[m*n];
    int *dArr;
    printf("Enter the elements of the matrix: ");
    for(int i=0; i<m*n; i++)
        scanf("%d", &arr[i]);
    hipMalloc(&dArr, m*n*sizeof(int));
    hipMemcpy(dArr, arr, m*n*sizeof(int), hipMemcpyHostToDevice);
    rowPower<<<m*n, 1>>>(dArr, m, n);
    hipMemcpy(arr, dArr, m*n*sizeof(int), hipMemcpyDeviceToHost);
    printf("Matrix:\n");
    for(int i=0; i<m*n; i++) {
        if(i%n == 0) printf("\n");
        printf("%d ", arr[i]);
    }
    hipFree(dArr);
    return 0;
}