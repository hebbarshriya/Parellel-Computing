#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void q1a(int*a, int*b, int* c, int wa, int wb){
	int ridA=threadIdx.x;
	int sum;
	for(int cidB=0;cidB<wb;cidB++){
		sum=0;
		for(int k=0;k<wa;k++){
			sum+=a[ridA*wa + k] * b[k*wb + cidB];
		}
		c[ridA*wb +cidB]=sum;
	}
}

__global__ void q1b(int*a, int*b, int* c ,int ha,int wa){
	int cidB=threadIdx.x;
	int wb=blockDim.x;
	int sum;
	for(int ridA=0;ridA<ha;ridA++){
		sum=0;
		for(int k=0;k<wa;k++){
			sum+=a[ridA*wa + k] * b[k*wb + cidB];
		}
		c[ridA*wb +cidB]=sum;
	}
}

__global__ void q1c(int*a, int*b, int* c ,int wa){
	int cidB=threadIdx.x;
	int ridA=threadIdx.y;
	int wb=blockDim.x;
	int sum=0;
	for(int k=0;k<wa;k++){
		sum+=a[ridA*wa + k] * b[k*wb + cidB];
	}
	c[ridA*wb +cidB]=sum;
	
}

int main(void)
{
	int *a,*b,*c,ha,wa,hb,wb,i,j;
	int *d_a,*d_b,*d_c;
	printf("Enter the value of ha and wa: ");
	scanf("%d %d",&ha,&wa);
	printf("Enter the value of wa and wb: ");
	scanf("%d %d",&hb,&wb);
	
	a=(int*)malloc(ha*wa*sizeof(int));
	b=(int*)malloc(hb*wb*sizeof(int));
	c=(int*)malloc(ha*wb*sizeof(int));

	printf("Enter input matrix A:\n");
	for(i=0;i<ha*wa;i++)
		scanf("%d",&a[i]);
	printf("Enter input matrix B:\n");
	for(i=0;i<hb*wb;i++)
		scanf("%d",&b[i]);

	hipMalloc((void**)&d_a,sizeof(int)*wa*ha);
	hipMalloc((void**)&d_b,sizeof(int)*wb*hb);
	hipMalloc((void**)&d_c,sizeof(int)*wb*ha);

	hipMemcpy(d_a,a,sizeof(int)*wa*ha,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,sizeof(int)*wb*hb,hipMemcpyHostToDevice);


	q1a<<<1,ha>>>(d_a,d_b,d_c,wa,wb);
	hipMemcpy(c,d_c,sizeof(int)*wb*ha,hipMemcpyDeviceToHost);
	printf("Result vector is:\n");
	for(i=0;i<ha;i++){
		for(j=0;j<wb;j++)
			printf("%d\t",c[i*ha+j]);
		printf("\n");
	}

	//q1b<<<1,wb>>>(d_a,d_b,d_c,ha,wa);
	//cudaMemcpy(c,d_c,sizeof(int)*wb*ha,cudaMemcpyDeviceToHost);
	//printf("Result vector is:\n");
	//for(i=0;i<ha;i++){
	//	for(j=0;j<wb;j++)
	//		printf("%d\t",c[i*ha+j]);
	//	printf("\n");
	//}

	//q1c<<<(1,1),(ha,wb)>>>(d_a,d_b,d_c,wa);
	//cudaMemcpy(c,d_c,sizeof(int)*wb*ha,cudaMemcpyDeviceToHost);
	//printf("Result vector is:\n");
	//for(i=0;i<ha;i++){
		//for(j=0;j<wb;j++)
		//	printf("%d\t",c[i*ha+j]);
		//printf("\n");
	//}

getchar();
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}